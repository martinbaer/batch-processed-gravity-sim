#include "hip/hip_runtime.h"
/**
 * @file bh_serial.cpp
 * @author Martin Baer
 * @brief 
 * Particle-particle simulator serial implimentation
 * This code simulates a given particle system for a given number of steps, 
 * saving the particle positions of each step to a file.
 * It also prints the time it took to run the simulation.
 * 
 * Barnes-Hut simulator serial implimentation
 * @version 0.1
 * @date 2022-11-01
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>
#include <limits>

#include "bh_tree.h"

#define USAGE "Usage: ./bh_serial [constants file]"
#define NUM_ARGS 2


void add_node_acceleration(double &acc_x, double &acc_y, double x, double y, unsigned int node_index, double s, BHTree bh_tree, Constants constants)
{
	Node node = bh_tree.nodes[node_index];
	// Calculate the distance between the particle and the node
	double dx = node.centre_of_mass_x - x;
	double dy = node.centre_of_mass_y - y;
	double d = sqrt(dx * dx + dy * dy);
	// If the node is a leaf, add the acceleration
	if (node.mass == 1)
	{
		// Calculate and add the acceleration (mass is 1)
		acc_x += dx / (d * d * d + constants.softening);
		acc_y += dy / (d * d * d + constants.softening);
	}
	// If the node is not a leaf, check if the node is far enough to take its centre of mass
	else
	{
		// Check the s/d ratio for the node
		if (s / d < constants.theta)
		{
			// Calculate and add the acceleration (mass is >1)
			acc_x += node.mass * dx / (d * d * d + constants.softening);
			acc_y += node.mass * dy / (d * d * d + constants.softening);
		}
		else
		{
			// Recursively calculate the acceleration
			double new_s = s / 2;
			if (node.bottom_left)
				add_node_acceleration(acc_x, acc_y, x, y, node.bottom_left, new_s, bh_tree, constants);
			if (node.bottom_right)
				add_node_acceleration(acc_x, acc_y, x, y, node.bottom_right, new_s, bh_tree, constants);
			if (node.top_left)
				add_node_acceleration(acc_x, acc_y, x, y, node.top_left, new_s, bh_tree, constants);
			if (node.top_right)
				add_node_acceleration(acc_x, acc_y, x, y, node.top_right, new_s, bh_tree, constants);
		}
	}
}


// add_node_acceleration but iterative instead of recursive
void add_node_acceleration_iterative(double &acc_x, double &acc_y, double x, double y, unsigned int node_index, double s, BHTree bh_tree, Constants constants)
{
	std::vector<unsigned int> nodes_to_check;
	nodes_to_check.push_back(node_index);
	while (nodes_to_check.size() > 0)
	{
		Node node = bh_tree.nodes[nodes_to_check.back()];
		nodes_to_check.pop_back();
		// Calculate the distance between the particle and the node
		double dx = node.centre_of_mass_x - x;
		double dy = node.centre_of_mass_y - y;
		double d = sqrt(dx * dx + dy * dy);
		// If the node is a leaf, add the acceleration
		if (node.mass == 1)
		{
			// Calculate and add the acceleration (mass is 1)
			acc_x += dx / (d * d * d + constants.softening);
			acc_y += dy / (d * d * d + constants.softening);
		}
		// If the node is not a leaf, check if the node is far enough to take its centre of mass
		else
		{
			// Check the s/d ratio for the node
			if (s / d < constants.theta)
			{
				// Calculate and add the acceleration (mass is >1)
				acc_x += node.mass * dx / (d * d * d + constants.softening);
				acc_y += node.mass * dy / (d * d * d + constants.softening);
			}
			else
			{
				// Recursively calculate the acceleration
				double new_s = s / 2;
				if (node.bottom_left)
					nodes_to_check.push_back(node.bottom_left);
				if (node.bottom_right)
					nodes_to_check.push_back(node.bottom_right);
				if (node.top_left)
					nodes_to_check.push_back(node.top_left);
				if (node.top_right)
					nodes_to_check.push_back(node.top_right);
			}
		}
	}
}


/**
 * @brief 
 * Simulates a particle system under the given constants 
 * and saves the particle positions to a binary file.
 */
int main(int argc, char *argv[])
{
	// start timer
	auto start = std::chrono::high_resolution_clock::now();

	// Check if the correct number of arguments were given
	if (argc != NUM_ARGS)
	{
		std::cerr << USAGE << std::endl;
		return 1;
	}
	// Parse the constants file
	Constants constants;
	parse_constants(argv[1], constants);
	// Check that the number of particles does not exceed the limitations of data types
	// (the BH tree must store pointers to the children of each node)
	if (2 * constants.num_particles > UINT_MAX)
	{
		std::cerr << "Error: Too many particles for BH tree: " << constants.num_particles << " * 2 > " << USHRT_MAX << std::endl;
		return 1;
	}
	// Initialise the output file
	std::ofstream output_file(constants.output_filename);
	// Check if the file opened
	if (!output_file.is_open())
	{
		std::cerr << "Error opening file " << argv[2] << std::endl;
		return 1;
	}
	// Initialise physical vectors
	std::vector<double> pos_x = std::vector<double>(constants.num_particles);
	std::vector<double> pos_y = std::vector<double>(constants.num_particles);
	std::vector<double> vel_x = std::vector<double>(constants.num_particles);
	std::vector<double> vel_y = std::vector<double>(constants.num_particles);
	std::vector<double> acc_x = std::vector<double>(constants.num_particles);
	std::vector<double> acc_y = std::vector<double>(constants.num_particles);

	// Initialise the BH tree
	BHTree bh_tree;
	bh_tree.nodes = new Node[constants.num_particles * 2];

	// Set the initial positions, velocities and accelerations
	for (int i = 0; i < constants.num_particles; i++)
	{
		pos_x[i] = constants.init_pos_x[i];
		pos_y[i] = constants.init_pos_y[i];
		vel_x[i] = 0;
		vel_y[i] = 0;
		acc_x[i] = 0;
		acc_y[i] = 0;
	}
	constants.init_pos_x.clear();
	constants.init_pos_y.clear();

	// Loop over the number of steps
	for (int step = 0; step < constants.num_steps; step++)
	{
		// Write the positions to the binary output file
		if (step % constants.write_interval == 0)
			write_positions(output_file, pos_x, pos_y, constants);
		// Check the energy conservation
		if (constants.log_energy_conservation) 
			log_energy_conservation(pos_x, pos_y, vel_x, vel_y, constants);

		// Calculate the bounds of the simulation : TODO AVX using _mm256_cmp_pd (and MPI)
		double min_x = pos_x[0];
		double max_x = pos_x[0];
		double min_y = pos_y[0];
		double max_y = pos_y[0];
		for (int i = 1; i < constants.num_particles; i++)
		{
			if (pos_x[i] < min_x)
				min_x = pos_x[i];
			else if (pos_x[i] > max_x)
				max_x = pos_x[i];
			if (pos_y[i] < min_y)
				min_y = pos_y[i];
			else if (pos_y[i] > max_y)
				max_y = pos_y[i];
		}

		// Create tree : TODO MPI by distrubiting particles, creating trees and merging them

		// Create root node info
		NodeDescriber root;
		root.centre_x = (max_x + min_x) / 2;
		root.centre_y = (max_y + min_y) / 2;
		root.half_width = max_x - min_x < max_y - min_y ? (max_y - min_y) / 2 : (max_x - min_x) / 2;
		// Reset the tree
		zero_node(bh_tree, ROOT_INDEX);
		bh_tree.num_nodes = 1;
		// Add each particle to the barnes-hut tree
		for (int i = 0; i < constants.num_particles; i++)
		{
			bh_tree_insert(pos_x[i], pos_y[i], bh_tree, root);
		}
		// Print tree
		// print_tree(0, bh_tree, TREE_ROOT);
		// exit(0);

		if (constants.log_tree_size)
			log_tree_size(bh_tree, constants);

		// Loop over each particle to calculate th_ acceleration: TODO CUDA
		for (int i = 0; i < constants.num_particles; i++)
		{
			// Get the acceleration for the particle
			add_node_acceleration(acc_x[i], acc_y[i], pos_x[i], pos_y[i], ROOT_INDEX, root.half_width, bh_tree, constants);
			acc_x[i] *= constants.gravity;
			acc_y[i] *= constants.gravity;
		}

		// Loop over the particles to update their velocities and positions
		for (int i = 0; i < constants.num_particles; i++)
		{
			// Update the velocity
			vel_x[i] += acc_x[i] * constants.delta_t;
			vel_y[i] += acc_y[i] * constants.delta_t;
			// Update the position
			pos_x[i] += vel_x[i] * constants.delta_t;
			pos_y[i] += vel_y[i] * constants.delta_t;
			// Reset the acceleration
			acc_x[i] = 0;
			acc_y[i] = 0;
		}
	}

	// Close output file
	output_file.close();
	// Free memory
	pos_x.clear();
	pos_y.clear();
	vel_x.clear();
	vel_y.clear();
	acc_x.clear();
	acc_y.clear();
	delete[] bh_tree.nodes;

	// Stop the timer
	auto end = std::chrono::high_resolution_clock::now();
	// Calculate the time taken
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	// Print the time taken
	std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

	return 0;
}