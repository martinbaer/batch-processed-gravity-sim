#include "hip/hip_runtime.h"
/**
 * @file bh_serial.cpp
 * @author Martin Baer
 * @brief 
 * Particle-particle simulator serial implimentation
 * This code simulates a given particle system for a given number of steps, 
 * saving the particle positions of each step to a file.
 * It also prints the time it took to run the simulation.
 * 
 * Barnes-Hut simulator serial implimentation
 * @version 0.1
 * @date 2022-11-01
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>
#include <limits>

#include "bh_tree.h"

#define USAGE "Usage: ./bh_serial [constants file]"
#define NUM_ARGS 2
#define BLOCK_SIZE 256

void add_node_acceleration(double &acc_x, double &acc_y, double x, double y, unsigned int node_index, double s, BHTree bh_tree, Constants constants)
{
	Node node = bh_tree.nodes[node_index];
	// Calculate the distance between the particle and the node
	double dx = node.centre_of_mass_x - x;
	double dy = node.centre_of_mass_y - y;
	double d = sqrt(dx * dx + dy * dy);
	// If the node is a leaf, add the acceleration
	if (node.mass == 1)
	{
		// Calculate and add the acceleration (mass is 1)
		acc_x += dx / (d * d * d + constants.softening);
		acc_y += dy / (d * d * d + constants.softening);
	}
	// If the node is not a leaf, check if the node is far enough to take its centre of mass
	else
	{
		// Check the s/d ratio for the node
		if (s / d < constants.theta)
		{
			// Calculate and add the acceleration (mass is >1)
			acc_x += node.mass * dx / (d * d * d + constants.softening);
			acc_y += node.mass * dy / (d * d * d + constants.softening);
		}
		else
		{
			// Recursively calculate the acceleration
			double new_s = s / 2;
			if (node.bottom_left)
				add_node_acceleration(acc_x, acc_y, x, y, node.bottom_left, new_s, bh_tree, constants);
			if (node.bottom_right)
				add_node_acceleration(acc_x, acc_y, x, y, node.bottom_right, new_s, bh_tree, constants);
			if (node.top_left)
				add_node_acceleration(acc_x, acc_y, x, y, node.top_left, new_s, bh_tree, constants);
			if (node.top_right)
				add_node_acceleration(acc_x, acc_y, x, y, node.top_right, new_s, bh_tree, constants);
		}
	}
}


__device__ add_node_acceleration_kernel(double &acc_x, double &acc_y, double x, double y, unsigned int node_index, double s, BHTree bh_tree, Constants constants)
{
	Node node = bh_tree.nodes[node_index];
	// Calculate the distance between the particle and the node
	double dx = node.centre_of_mass_x - x;
	double dy = node.centre_of_mass_y - y;
	double d = sqrt(dx * dx + dy * dy);
	// If the node is a leaf, add the acceleration
	if (node.mass == 1)
	{
		// Calculate and add the acceleration (mass is 1)
		acc_x += dx / (d * d * d + constants.softening);
		acc_y += dy / (d * d * d + constants.softening);
	}
	// If the node is not a leaf, check if the node is far enough to take its centre of mass
	else
	{
		// Check the s/d ratio for the node
		if (s / d < constants.theta)
		{
			// Calculate and add the acceleration (mass is >1)
			acc_x += node.mass * dx / (d * d * d + constants.softening);
			acc_y += node.mass * dy / (d * d * d + constants.softening);
		}
		else
		{
			// Recursively calculate the acceleration
			double new_s = s / 2;
			if (node.bottom_left)
				add_node_acceleration_kernel(acc_x, acc_y, x, y, node.bottom_left, new_s, bh_tree, constants);
			if (node.bottom_right)
				add_node_acceleration_kernel(acc_x, acc_y, x, y, node.bottom_right, new_s, bh_tree, constants);
			if (node.top_left)
				add_node_acceleration_kernel(acc_x, acc_y, x, y, node.top_left, new_s, bh_tree, constants);
			if (node.top_right)
				add_node_acceleration_kernel(acc_x, acc_y, x, y, node.top_right, new_s, bh_tree, constants);
		}
	}
}

// Calculate the node accleration and then multiply it by gravity
__device__ calculate_acceleration_kernel(ArrayVector2D pos, ArrayVector2D acc, BHTree bh_tree, Constants constants)
{
	// starting index for the thread's row
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// Zero the acceleration
	acc[i].x = 0;
	acc[i].y = 0;
	// Calculate the acceleration for the particle using iteration instead of recursion
	
	// Multiply by gravity
	acc[i].x *= constants.gravity;
	acc[i].y *= constants.gravity;
}


/**
 * @brief 
 * Simulates a particle system under the given constants 
 * and saves the particle positions to a binary file.
 */
int main(int argc, char *argv[])
{
	// start timer
	auto start = std::chrono::high_resolution_clock::now();

	// Check if the correct number of arguments were given
	if (argc != NUM_ARGS)
	{
		std::cerr << USAGE << std::endl;
		return 1;
	}
	// Parse the constants file
	Constants constants;
	parse_constants(argv[1], constants);
	// Initialise the output file
	std::ofstream output_file(constants.output_filename);
	// Check if the file opened
	if (!output_file.is_open())
	{
		std::cerr << "Error opening file " << argv[2] << std::endl;
		return 1;
	}

	// Initialise physical vectors
	ArrayVector2D pos;
	ArrayVector2D vel;
	ArrayVector2D acc;
	pos.x = new double[constants.num_particles];
	pos.y = new double[constants.num_particles];
	vel.x = new double[constants.num_particles];
	vel.y = new double[constants.num_particles];
	acc.x = new double[constants.num_particles];
	acc.y = new double[constants.num_particles];
	// Initialise physical vectors on the GPU
	ArrayVector2D pos_device;
	ArrayVector2D vel_device;
	ArrayVector2D acc_device;
	checkError(hipMalloc(&pos_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&pos_device.y, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&vel_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&vel_device.y, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&acc_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&acc_device.y, constants.num_particles * sizeof(double)));

	// Initialise the BH tree
	BHTree bh_tree;
	bh_tree.max_nodes = constants.num_particles * 2;
	bh_tree.nodes = new Node[bh_tree.max_nodes];
	// Initialise the BH tree on the GPU
	BHTree bh_tree_device;
	checkError(hipMalloc(&bh_tree_device.nodes, bh_tree.max_nodes * sizeof(Node)));

	// Set the initial positions, velocities and accelerations
	for (int i = 0; i < constants.num_particles; i++)
	{
		pos.x[i] = constants.init_pos.x[i];
		pos.y[i] = constants.init_pos.y[i];
		vel.x[i] = 0;
		vel.y[i] = 0;
		acc.x[i] = 0;
		acc.y[i] = 0;
	}
	// Delete the initial positions in the constants struct
	delete[] constants.init_pos.x;
	delete[] constants.init_pos.y;

	// Loop over the number of steps
	for (int step = 0; step < constants.num_steps; step++)
	{
		// Write the positions to the binary output file
		if (step % constants.write_interval == 0)
			write_positions(output_file, pos, constants);
		// Check the energy conservation
		if (constants.log_energy_conservation) 
			log_energy_conservation(pos, vel, constants);

		// Calculate the bounds of the simulation : TODO AVX using _mm256_cmp_pd (and MPI)
		double min_x = pos.x[0];
		double max_x = pos.x[0];
		double min_y = pos.y[0];
		double max_y = pos.y[0];
		for (int i = 1; i < constants.num_particles; i++)
		{
			if (pos.x[i] < min_x)
				min_x = pos.x[i];
			else if (pos.x[i] > max_x)
				max_x = pos.x[i];
			if (pos.y[i] < min_y)
				min_y = pos.y[i];
			else if (pos.y[i] > max_y)
				max_y = pos.y[i];
		}

		// Create tree : TODO MPI by distrubiting particles, creating trees and merging them

		// Create root node info
		NodeDescriber root;
		root.index = ROOT_INDEX;
		root.centre_x = (max_x + min_x) / 2;
		root.centre_y = (max_y + min_y) / 2;
		root.half_width = max_x - min_x < max_y - min_y ? (max_y - min_y) / 2 : (max_x - min_x) / 2;
		// Reset the tree
		zero_node(bh_tree, ROOT_INDEX);
		bh_tree.num_nodes = 1;
		// Add each particle to the barnes-hut tree
		for (int i = 0; i < constants.num_particles; i++)
		{
			bh_tree_insert(pos.x[i], pos.y[i], bh_tree, root);
		}
		// Print tree
		// print_tree(0, bh_tree, TREE_ROOT);
		// exit(0);

		if (constants.log_tree_size)
			log_tree_size(bh_tree, constants);



		/* Phase 2: Calculate accleration */


		// Loop over each particle to calculate th_ acceleration: TODO CUDA

		// Copy the tree to the GPU

		// Reallocate the tree on the GPU if it is too small (this will only happen a couple of times)
		if (bh_tree_device.max_nodes < bh_tree.num_nodes)
		{
			checkError(hipFree(bh_tree_device.nodes));
			checkError(hipMalloc(&bh_tree_device.nodes, bh_tree.num_nodes * sizeof(Node)));
			bh_tree_device.max_nodes = bh_tree.num_nodes;
		}

		// Copy the tree to the GPU
		checkError(hipMemcpy(bh_tree_device.nodes, bh_tree.nodes, bh_tree.max_nodes * sizeof(Node), hipMemcpyHostToDevice));

		// Call the CUDA kernel to calculate the acceleration
		calculate_acceleration_kernel<<<constants.num_particles / BLOCK_SIZE + 1, BLOCK_SIZE>>>(pos_device, acc_device, bh_tree_device, constants);

		// for (int i = 0; i < constants.num_particles; i++)
		// {
		// 	// Get the acceleration for the particle
		// 	calc_node_acceleration(acc.x[i], acc.y[i], pos.x[i], pos.y[i], ROOT_INDEX, root.half_width, bh_tree, constants);
		// 	acc.x[i] *= constants.gravity;
		// 	acc.y[i] *= constants.gravity;
		// }

		// Loop over the particles to update their velocities and positions
		for (int i = 0; i < constants.num_particles; i++)
		{
			// Update the velocity
			vel.x[i] += acc.x[i] * constants.delta_t;
			vel.y[i] += acc.y[i] * constants.delta_t;
			// Update the position
			pos.x[i] += vel.x[i] * constants.delta_t;
			pos.y[i] += vel.y[i] * constants.delta_t;
			// Reset the acceleration
			acc.x[i] = 0;
			acc.y[i] = 0;
		}
	}

	// Close output file
	output_file.close();
	// Free memory
	delete[] pos.x;
	delete[] pos.y;
	delete[] vel.x;
	delete[] vel.y;
	delete[] acc.x;
	delete[] acc.y;
	delete[] bh_tree.nodes;
	// Free memory on the GPU
	checkError(hipFree(pos_device.x));
	checkError(hipFree(pos_device.y));
	checkError(hipFree(vel_device.x));
	checkError(hipFree(vel_device.y));
	checkError(hipFree(acc_device.x));
	checkError(hipFree(acc_device.y));
	checkError(hipFree(bh_tree_device.nodes));

	// Stop the timer
	auto end = std::chrono::high_resolution_clock::now();
	// Calculate the time taken
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	// Print the time taken
	std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

	return 0;
}