#include "hip/hip_runtime.h"
/**
 * @file bh_serial.cpp
 * @author Martin Baer
 * @brief 
 * Particle-particle simulator serial implimentation
 * This code simulates a given particle system for a given number of steps, 
 * saving the particle positions of each step to a file.
 * It also prints the time it took to run the simulation.
 * 
 * Barnes-Hut simulator serial implimentation
 * @version 0.1
 * @date 2022-11-01
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>
#include <limits>
#include <iomanip>
#include <sstream>
#include <chrono>

#include "bh_tree.h"

#define USAGE "Usage: ./bh_serial [constants file]"
#define NUM_ARGS 2
#define BLOCK_SIZE 256

#define QUEUE_EXPANSION_FACTOR 1.5

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}


typedef struct QueuedNode
{
   unsigned int tree_index;
   double s;
} QueuedNode;

typedef struct CircularArrayQueue
{
   QueuedNode *data;
   int num_elements;
   int max_elements;
   int head;
   int tail;
} CircularArrayQueue;

__device__ void enqueue(CircularArrayQueue &queue, unsigned int tree_index, double s)
{
	int new_tail = (queue.tail + 1) % queue.max_elements;
	// Expand the queue if needed (this should only occur a few times)
	if (new_tail == queue.head)
	{
		// Copy the queue data to a temporary array
		QueuedNode *temp_data = new QueuedNode[queue.max_elements];
		for (int i = 0; i < queue.max_elements; i++)
		{
			temp_data[i] = queue.data[i];
		}
		// Delete the queue data
		delete[] queue.data;
		// Reallocate the queue
		queue.max_elements = queue.max_elements * QUEUE_EXPANSION_FACTOR;
		queue.data = new QueuedNode[queue.max_elements];
		// Copy the data back from the temporary array
		for (int i = 0; i < queue.max_elements; i++)
		{
			queue.data[i] = temp_data[i];
		}
		// Delete the temporary array
		delete[] temp_data;
	}
	// Add the value to the queue
	queue.data[new_tail].tree_index = tree_index;
	queue.data[new_tail].s = s;
	queue.tail = new_tail;
	queue.num_elements++;
}

__device__ QueuedNode dequeue(CircularArrayQueue &queue)
{
	// Get the value at the head of the queue
	QueuedNode head = queue.data[queue.head];
	// Move the head of the queue
	queue.head = (queue.head - 1) % queue.max_elements;
	queue.num_elements--;
	return head;
}

// Calculate the node accleration and then multiply it by gravity 
__global__ void calculate_acceleration_kernel(ArrayVector2D pos, ArrayVector2D acc, BHTree bh_tree, double root_half_width, Constants constants)
{
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < constants.num_particles)
		return;
	// Create queue
	CircularArrayQueue queue;
	int expected_size = (int)__log2f(bh_tree.num_nodes);
	queue.data = new QueuedNode[expected_size];
	queue.num_elements = 0;
	queue.max_elements = expected_size;
	queue.head = 0;
	queue.tail = 0;
	// Add the root node to the queue
	enqueue(queue, ROOT_INDEX, root_half_width);
	// Add node acceleration to the acceleration of the particle
	while (queue.num_elements > 0)
	{
		// Get the next node from the queue
		QueuedNode node = dequeue(queue);
		// Calculate the distance between the particle and the node
		double dx = bh_tree.nodes[node.tree_index].centre_of_mass_x - pos.x[thread_id];
		double dy = bh_tree.nodes[node.tree_index].centre_of_mass_y - pos.y[thread_id];
		double d = sqrt(dx * dx + dy * dy);
		// If the node is a leaf, add the acceleration
		if (bh_tree.nodes[node.tree_index].mass == 1)
		{
			// Calculate and add the acceleration (mass is 1)
			acc.x[thread_id] += constants.gravity * dx / (d * d * d + constants.softening);
			acc.y[thread_id] += constants.gravity * dy / (d * d * d + constants.softening);
		}
		else // Check if the node is far enough to take its centre of mass
		{
			// Check the s/d ratio for the node
			if (node.s / d < constants.theta)
			{
				// Calculate and add the acceleration
				acc.x[thread_id] += constants.gravity * bh_tree.nodes[node.tree_index].mass * dx / (d * d * d + constants.softening);
				acc.y[thread_id] += constants.gravity * bh_tree.nodes[node.tree_index].mass * dy / (d * d * d + constants.softening);
			}
			else // Add the children to the queue
			{
				// Add the children to the queue
				double child_s = node.s / 2;
				if (bh_tree.nodes[node.tree_index].bottom_left)
					enqueue(queue, bh_tree.nodes[node.tree_index].bottom_left, child_s);
				if (bh_tree.nodes[node.tree_index].bottom_right)
					enqueue(queue, bh_tree.nodes[node.tree_index].bottom_right, child_s);
				if (bh_tree.nodes[node.tree_index].top_left)
					enqueue(queue, bh_tree.nodes[node.tree_index].top_left, child_s);
				if (bh_tree.nodes[node.tree_index].top_right)
					enqueue(queue, bh_tree.nodes[node.tree_index].top_right, child_s);
			}
		}
	}
	// Multiply the acceleration by gravity
	acc.x[thread_id] *= constants.gravity;
	acc.y[thread_id] *= constants.gravity;
}


/**
 * @brief 
 * Simulates a particle system under the given constants 
 * and saves the particle positions to a binary file.
 */
int main(int argc, char *argv[])
{
	// start timer
	auto start = std::chrono::high_resolution_clock::now();

	// Check if the correct number of arguments were given
	if (argc != NUM_ARGS)
	{
		std::cerr << USAGE << std::endl;
		return 1;
	}
	// Parse the constants file
	Constants constants;
	parse_constants(argv[1], constants);
	// Initialise the output file
	std::ofstream output_file(constants.output_filename);
	// Check if the file opened
	if (!output_file.is_open())
	{
		std::cerr << "Error opening file: " << argv[2] << std::endl;
		return 1;
	}

	// Initialise physical vectors
	ArrayVector2D pos;
	ArrayVector2D vel;
	ArrayVector2D acc;
	pos.x = new double[constants.num_particles];
	pos.y = new double[constants.num_particles];
	vel.x = new double[constants.num_particles];
	vel.y = new double[constants.num_particles];
	acc.x = new double[constants.num_particles];
	acc.y = new double[constants.num_particles];
	// Initialise physical vectors on the GPU
	ArrayVector2D pos_device;
	ArrayVector2D vel_device;
	ArrayVector2D acc_device;

	checkError(hipMalloc(&pos_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&pos_device.y, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&vel_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&vel_device.y, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&acc_device.x, constants.num_particles * sizeof(double)));
	checkError(hipMalloc(&acc_device.y, constants.num_particles * sizeof(double)));

	// Initialise the BH tree
	BHTree bh_tree;
	bh_tree.max_nodes = constants.num_particles * 2;
	bh_tree.nodes = new Node[bh_tree.max_nodes];
	// Initialise the BH tree on the GPU
	BHTree bh_tree_device;
	checkError(hipMalloc(&bh_tree_device.nodes, bh_tree.max_nodes * sizeof(Node)));

	// Set the initial positions, velocities and accelerations
	for (int i = 0; i < constants.num_particles; i++)
	{
		pos.x[i] = constants.init_pos.x[i];
		pos.y[i] = constants.init_pos.y[i];
		vel.x[i] = 0;
		vel.y[i] = 0;
		acc.x[i] = 0;
		acc.y[i] = 0;
	}
	// Delete the initial positions in the constants struct
	delete[] constants.init_pos.x;
	delete[] constants.init_pos.y;

	// Loop over the number of steps
	for (int step = 0; step < constants.num_steps; step++)
	{
		// Write the positions to the binary output file
		if (step % constants.write_interval == 0)
			write_positions(output_file, pos, constants);
		// Check the energy conservation
		if (constants.log_energy_conservation) 
			log_energy_conservation(pos, vel, constants);

		// Calculate the bounds of the simulation : TODO AVX using _mm256_cmp_pd (and MPI)
		double min_x = pos.x[0];
		double max_x = pos.x[0];
		double min_y = pos.y[0];
		double max_y = pos.y[0];
		for (int i = 1; i < constants.num_particles; i++)
		{
			if (pos.x[i] < min_x)
				min_x = pos.x[i];
			else if (pos.x[i] > max_x)
				max_x = pos.x[i];
			if (pos.y[i] < min_y)
				min_y = pos.y[i];
			else if (pos.y[i] > max_y)
				max_y = pos.y[i];
		}

		// Create tree : TODO MPI by distrubiting particles, creating trees and merging them

		// Create root node info
		NodeDescriber root;
		root.index = ROOT_INDEX;
		root.centre_x = (max_x + min_x) / 2;
		root.centre_y = (max_y + min_y) / 2;
		root.half_width = max_x - min_x < max_y - min_y ? (max_y - min_y) / 2 : (max_x - min_x) / 2;
		// Reset the tree
		zero_node(bh_tree, ROOT_INDEX);
		bh_tree.num_nodes = 1;
		// Add each particle to the barnes-hut tree
		for (int i = 0; i < constants.num_particles; i++)
		{
			bh_tree_insert(pos.x[i], pos.y[i], bh_tree, root);
		}
		// Print tree
		// print_tree(0, bh_tree, TREE_ROOT);
		// exit(0);

		if (constants.log_tree_size)
			log_tree_size(bh_tree, constants);



		/* Phase 2: Calculate accleration */

		// Reallocate the tree on the GPU if it is too small (this will only happen a couple of times)
		if (bh_tree_device.max_nodes < bh_tree.num_nodes)
		{
			checkError(hipFree(bh_tree_device.nodes));
			checkError(hipMalloc(&bh_tree_device.nodes, bh_tree.num_nodes * sizeof(Node)));
			bh_tree_device.max_nodes = bh_tree.num_nodes;
		}
		// Copy the tree to the GPU
		checkError(hipMemcpy(bh_tree_device.nodes, bh_tree.nodes, bh_tree.max_nodes * sizeof(Node), hipMemcpyHostToDevice));

		// Copy the positions to the GPU
		checkError(hipMemcpy(pos_device.x, pos.x, constants.num_particles * sizeof(double), hipMemcpyHostToDevice));
		checkError(hipMemcpy(pos_device.y, pos.y, constants.num_particles * sizeof(double), hipMemcpyHostToDevice));

		// Call the CUDA kernel to calculate the acceleration pos_device.x, pos_device.y, acc_device.x, acc_device.y, bh_tree_device.nodes, root.half_width, 
		calculate_acceleration_kernel<<<constants.num_particles / BLOCK_SIZE + 1, BLOCK_SIZE>>>(pos_device, acc_device, bh_tree_device, root.half_width, constants);


		// Loop over the particles to update their velocities and positions
		for (int i = 0; i < constants.num_particles; i++)
		{
			// Update the velocity
			vel.x[i] += acc.x[i] * constants.delta_t;
			vel.y[i] += acc.y[i] * constants.delta_t;
			// Update the position
			pos.x[i] += vel.x[i] * constants.delta_t;
			pos.y[i] += vel.y[i] * constants.delta_t;
			// Reset the acceleration
			acc.x[i] = 0;
			acc.y[i] = 0;
		}
	}

	// Close output file
	output_file.close();
	// Free memory
	delete[] pos.x;
	delete[] pos.y;
	delete[] vel.x;
	delete[] vel.y;
	delete[] acc.x;
	delete[] acc.y;
	delete[] bh_tree.nodes;
	// Free memory on the GPU
	checkError(hipFree(pos_device.x));
	checkError(hipFree(pos_device.y));
	checkError(hipFree(vel_device.x));
	checkError(hipFree(vel_device.y));
	checkError(hipFree(acc_device.x));
	checkError(hipFree(acc_device.y));
	checkError(hipFree(bh_tree_device.nodes));

	// Stop the timer
	auto end = std::chrono::high_resolution_clock::now();
	// Calculate the time taken
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	// Print the time taken
	std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

	return 0;
}